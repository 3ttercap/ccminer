#include "hip/hip_runtime.h"
/**
 * Optimized Blake-256 8-rounds Cuda Kernel (Tested on SM >3.0)
 * Based upon Blake-256 implementation of Tanguy Pruvot - Nov. 2014
 *
 * midstate computation inherited from
 *  https://github.com/wfr/clblake
 *
 * Provos Alexis - Jan. 2016
 * Reviewed by tpruvot - Feb 2016
 */

#include <stdint.h>
#include <memory.h>
#include <emmintrin.h>

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
}

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

/* threads per block and "magic" */
#define TPB 768
#define NPT 224
#define NBN 2

__constant__ uint32_t d_data[16];

/* 8 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

/* hash by cpu with blake 256 */
extern "C" void blake256hash(void *output, const void *input, int8_t blakerounds)
{
	uchar hash[64];
	sph_blake256_context ctx;

	sph_blake256_set_rounds(blakerounds);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);

	memcpy(output, hash, 32);
}

__global__ __launch_bounds__(TPB,1)
void vanilla_gpu_hash_16_8(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,const uint32_t highTarget){
	uint32_t v[16];
	uint32_t tmp[13];

	const uint32_t thread   = blockDim.x * blockIdx.x + threadIdx.x;
	const uint32_t step     = gridDim.x * blockDim.x;
	const uint32_t maxNonce = startNonce + threads;

	const	uint32_t c_u256[16] = {
				  0x243F6A88,	0x85A308D3,	0x13198A2E,	0x03707344,
				  0xA4093822,	0x299F31D0,	0x082EFA98,	0xEC4E6C89,
				  0x452821E6,	0x38D01377,	0xBE5466CF,	0x34E90C6C,
				  0xC0AC29B7,	0xC97C50DD,	0x3F84D5B5,	0xB5470917
	};
	const	uint32_t h[8]  = {
				  d_data[0],	d_data[1],	d_data[2],	d_data[3],
				  d_data[4],	0,		d_data[5],	d_data[6]
	};
		uint32_t m[16] = {
				  d_data[7],	d_data[8],	d_data[9],	0,
				  0x80000000UL,	0,		0,		0,
				  0,		0,		0,		0,
				  0,		1,		0,		640
		};

	#pragma unroll 6
	for(int i=0;i<6;i++)
		tmp[ i] = d_data[i+10U];
		
	//---MORE PRECOMPUTATIONS
	tmp[ 6] = c_u256[2] + tmp[ 4];	tmp[ 7] = c_u256[1] + tmp[ 2];

	tmp[ 4] = __byte_perm(tmp[ 4] ^ h[2],0, 0x0321);	tmp[ 6] += tmp[ 4];
	tmp[ 5] = ROTR32(tmp[ 5] ^ tmp[ 6],7);			tmp[ 8] = __byte_perm(c_u256[7] ^ h[3],0, 0x1032);
	tmp[ 9] = c_u256[3] + tmp[8];                   	tmp[10] = ROTR32(h[7] ^ tmp[9], 12);
	tmp[11] = h[3] + c_u256[6] + tmp[10];

	tmp[ 8] = __byte_perm(tmp[8] ^ tmp[11],0, 0x0321);  tmp[ 9] += tmp[8];
	tmp[10] = ROTR32(tmp[10] ^ tmp[9],7);
	//---END OF MORE PRECOMPUTATIONS

	for(uint64_t m3 = startNonce + thread ; m3<maxNonce ; m3+=step){

		m[3]  = m3;

		//All i need is, h0,h1,h2,h4,h6,h7,m0,m1,m2 ++ tmps (13) //22 vars
		v[ 0] = h[ 0];  	v[ 1] = h[1];		v[ 2] = h[2];		v[ 3] = tmp[11];
		v[ 4] = h[ 4];  	v[ 5] = tmp[ 3];	v[ 6] = tmp[ 5];	v[ 7] = tmp[10];
		v[ 8] = tmp[ 1];	v[ 9] = tmp[ 7];	v[10] = tmp[ 6];	v[11] = tmp[ 9];
		v[12] = tmp[ 0];	v[13] = tmp[ 2];	v[14] = tmp[ 4];	v[15] = tmp[ 8];

		v[ 1] += m3 ^ c_u256[2];        v[13] = __byte_perm(v[13] ^ v[1],0, 0x0321);v[ 9] += v[13];     v[5] = ROTR32(v[5] ^ v[9], 7);
		v[ 0] += v[5];                  v[15] = __byte_perm(v[15] ^ v[0],0, 0x1032);v[10] += v[15];     v[5] = ROTR32(v[5] ^ v[10], 12);
		v[ 0] += c_u256[8] + v[5];      v[15] = __byte_perm(v[15] ^ v[0],0, 0x0321);v[10] += v[15];     v[5] = ROTR32(v[5] ^ v[10], 7);

		#define GSPREC(a,b,c,d,x,y) { \
			v[a] += (m[x] ^ c_u256[y]) + v[b]; \
			v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
			v[c] += v[d]; \
			v[b] = ROTR32(v[b] ^ v[c], 12); \
			v[a] += (m[y] ^ c_u256[x]) + v[b]; \
			v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
			v[c] += v[d]; \
			v[b] = ROTR32(v[b] ^ v[c], 7); \
		}

						GSPREC(1, 6, 11, 12, 10, 11);   GSPREC(2, 7, 8, 13, 12, 13);    GSPREC(3, 4, 9, 14, 14, 15);
		//  { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 8, 12, 14, 10);    GSPREC(1, 5, 9, 13, 4, 8);      GSPREC(2, 6, 10, 14, 9, 15);    GSPREC(3, 7, 11, 15, 13, 6);
		GSPREC(0, 5, 10, 15, 1, 12);    GSPREC(1, 6, 11, 12, 0, 2);     GSPREC(2, 7, 8, 13, 11, 7);     GSPREC(3, 4, 9, 14, 5, 3);
		//  { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 8, 12, 11, 8);     GSPREC(1, 5, 9, 13, 12, 0);     GSPREC(2, 6, 10, 14, 5, 2);     GSPREC(3, 7, 11, 15, 15, 13);
		GSPREC(0, 5, 10, 15, 10, 14);   GSPREC(1, 6, 11, 12, 3, 6);     GSPREC(2, 7, 8, 13, 7, 1);      GSPREC(3, 4, 9, 14, 9, 4);
		//  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 8, 12, 7, 9);      GSPREC(1, 5, 9, 13, 3, 1);      GSPREC(2, 6, 10, 14, 13, 12);   GSPREC(3, 7, 11, 15, 11, 14);
		GSPREC(0, 5, 10, 15, 2, 6);     GSPREC(1, 6, 11, 12, 5, 10);    GSPREC(2, 7, 8, 13, 4, 0);      GSPREC(3, 4, 9, 14, 15, 8);
		//  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 8, 12, 9, 0);      GSPREC(1, 5, 9, 13, 5, 7);      GSPREC(2, 6, 10, 14, 2, 4);     GSPREC(3, 7, 11, 15, 10, 15);
		GSPREC(0, 5, 10, 15, 14, 1);    GSPREC(1, 6, 11, 12, 11, 12);   GSPREC(2, 7, 8, 13, 6, 8);      GSPREC(3, 4, 9, 14, 3, 13);
		//  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 8, 12, 2, 12);     GSPREC(1, 5, 9, 13, 6, 10);     GSPREC(2, 6, 10, 14, 0, 11);    GSPREC(3, 7, 11, 15, 8, 3);
		GSPREC(0, 5, 10, 15, 4, 13);    GSPREC(1, 6, 11, 12, 7, 5);     GSPREC(2, 7, 8, 13, 15, 14);    GSPREC(3, 4, 9, 14, 1, 9);
		//  { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 8, 12, 12, 5);     GSPREC(1, 5, 9, 13, 1, 15);     GSPREC(2, 6, 10, 14, 14, 13);   GSPREC(3, 7, 11, 15, 4, 10);
		GSPREC(0, 5, 10, 15, 0, 7);     GSPREC(1, 6, 11, 12, 6, 3);     GSPREC(2, 7, 8, 13, 9, 2);      GSPREC(3, 4, 9, 14, 8, 11);
		//  { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 8, 12, 13, 11);    GSPREC(1, 5, 9, 13, 7, 14);     GSPREC(2, 6, 10, 14, 12, 1);    GSPREC(3, 7, 11, 15, 3, 9);

		v[ 0] += (m[ 5] ^ c_u256[0]) + v[5];	v[15] = __byte_perm(v[15] ^ v[0],0, 0x1032);
		v[10] += v[15];				v[ 5] = ROTR32(v[5] ^ v[10], 12);
		v[ 0] += (m[ 0] ^ c_u256[5]) + v[5];	v[15] = __byte_perm(v[15] ^ v[0],0, 0x0321);

		v[2] += (m[ 8] ^ c_u256[6]) + v[7];	v[13] = __byte_perm(v[13] ^ v[2],0, 0x1032);
		v[8] += v[13];				v[ 7] = ROTR32(v[7] ^ v[8], 12);
		v[2] += (m[ 6] ^ c_u256[8]) + v[7];	v[13] = __byte_perm(v[13] ^ v[2],0, 0x0321);
		v[8] += v[13];				v[ 7] = ROTR32(v[7] ^ v[8], 7);

		// only compute h6 & 7
		if((h[7]^v[7]^v[15])==0){
			GSPREC(1, 6, 11, 12, 15, 4);
			v[ 3] += (m[2] ^ c_u256[10]) + v[4];
			v[14]  = __byte_perm(v[14] ^ v[3],0, 0x1032);
			v[ 9] += v[14];
			v[ 4]  = ROTR32(v[4] ^ v[9],12);
			v[ 3] += (m[10] ^ c_u256[2]) + v[4];
			v[14]  = __byte_perm(v[14] ^ v[3],0, 0x0321);
			if(cuda_swab32(h[6]^v[6]^v[14]) <= highTarget) {
#if NBN == 2
			/* keep the smallest nonce, + extra one if found */
			if (m3 < resNonce[0]){
				resNonce[1] = resNonce[0];
				resNonce[0] = m3;
			}
			else
				resNonce[1] = m3;
#else
			resNonce[0] = m3;
#endif
			}
		}
	}
}


#define round(r) \
		/*        column step          */ \
		buf1 = _mm_set_epi32(m.u32[sig[r][ 6]], m.u32[sig[r][ 4]], m.u32[sig[r][ 2]], m.u32[sig[r][ 0]]); \
		buf2  = _mm_set_epi32(z[sig[r][ 7]], z[sig[r][ 5]], z[sig[r][ 3]],z[sig[r][ 1]]); \
		buf1 = _mm_xor_si128( buf1, buf2); \
		row1 = _mm_add_epi32( _mm_add_epi32( row1, buf1), row2 ); \
		buf1  = _mm_set_epi32(z[sig[r][ 6]], z[sig[r][ 4]], z[sig[r][ 2]], z[sig[r][ 0]]); \
		buf2 = _mm_set_epi32(m.u32[sig[r][ 7]], m.u32[sig[r][ 5]], m.u32[sig[r][ 3]], m.u32[sig[r][ 1]]); \
		row4 = _mm_xor_si128( row4, row1 ); \
		row4 = _mm_xor_si128(_mm_srli_epi32( row4, 16 ),_mm_slli_epi32( row4, 16 )); \
		row3 = _mm_add_epi32( row3, row4 );   \
		row2 = _mm_xor_si128( row2, row3 ); \
		buf1 = _mm_xor_si128( buf1, buf2); \
		row2 = _mm_xor_si128(_mm_srli_epi32( row2, 12 ),_mm_slli_epi32( row2, 20 )); \
		row1 = _mm_add_epi32( _mm_add_epi32( row1, buf1), row2 ); \
		row4 = _mm_xor_si128( row4, row1 ); \
		row4 = _mm_xor_si128(_mm_srli_epi32( row4,  8 ),_mm_slli_epi32( row4, 24 )); \
		row3 = _mm_add_epi32( row3, row4 );   \
		row4 = _mm_shuffle_epi32( row4, _MM_SHUFFLE(2,1,0,3) ); \
		row2 = _mm_xor_si128( row2, row3 ); \
		row2 = _mm_xor_si128(_mm_srli_epi32( row2,  7 ),_mm_slli_epi32( row2, 25 )); \
\
		row3 = _mm_shuffle_epi32( row3, _MM_SHUFFLE(1,0,3,2) ); \
		row2 = _mm_shuffle_epi32( row2, _MM_SHUFFLE(0,3,2,1) ); \
\
	   /*       diagonal step         */ \
		buf1 = _mm_set_epi32(m.u32[sig[r][14]], m.u32[sig[r][12]], m.u32[sig[r][10]], m.u32[sig[r][ 8]]); \
		buf2  = _mm_set_epi32(z[sig[r][15]], z[sig[r][13]], z[sig[r][11]], z[sig[r][ 9]]); \
		buf1 = _mm_xor_si128( buf1, buf2); \
		row1 = _mm_add_epi32( _mm_add_epi32( row1, buf1 ), row2 ); \
		buf1  = _mm_set_epi32(z[sig[r][14]], z[sig[r][12]], z[sig[r][10]], z[sig[r][ 8]]); \
		buf2 = _mm_set_epi32(m.u32[sig[r][15]], m.u32[sig[r][13]], m.u32[sig[r][11]], m.u32[sig[r][ 9]]); \
		row4 = _mm_xor_si128( row4, row1 ); \
		buf1 = _mm_xor_si128( buf1, buf2); \
		row4 = _mm_xor_si128(_mm_srli_epi32( row4, 16 ),_mm_slli_epi32( row4, 16 )); \
		row3 = _mm_add_epi32( row3, row4 );   \
		row2 = _mm_xor_si128( row2, row3 ); \
		row2 = _mm_xor_si128(_mm_srli_epi32( row2, 12 ),_mm_slli_epi32( row2, 20 )); \
		row1 = _mm_add_epi32( _mm_add_epi32( row1, buf1 ), row2 ); \
		row4 = _mm_xor_si128( row4, row1 ); \
		row4 = _mm_xor_si128(_mm_srli_epi32( row4,  8 ),_mm_slli_epi32( row4, 24 )); \
		row3 = _mm_add_epi32( row3, row4 );   \
		row4 = _mm_shuffle_epi32( row4, _MM_SHUFFLE(0,3,2,1) ); \
		row2 = _mm_xor_si128( row2, row3 ); \
		row2 = _mm_xor_si128(_mm_srli_epi32( row2,  7 ),_mm_slli_epi32( row2, 25 )); \
\
		row3 = _mm_shuffle_epi32( row3, _MM_SHUFFLE(1,0,3,2) ); \
		row2 = _mm_shuffle_epi32( row2, _MM_SHUFFLE(2,1,0,3) ); \
\

#define LOADU(p)  _mm_loadu_si128( (__m128i *)(p) )

#define BSWAP32(r) do{ \
   r = _mm_shufflehi_epi16(r, _MM_SHUFFLE(2, 3, 0, 1));\
   r = _mm_shufflelo_epi16(r, _MM_SHUFFLE(2, 3, 0, 1));\
   r = _mm_xor_si128(_mm_slli_epi16(r, 8), _mm_srli_epi16(r, 8));\
} while(0)


__host__
void vanilla_cpu_setBlock_16(const uint32_t* endiandata, uint32_t *penddata){

	uint32_t _ALIGN(32) h[16];
	h[0]=0x6A09E667;    h[1]=0xBB67AE85;    h[2]=0x3C6EF372;    h[3]=0xA54FF53A;
	h[4]=0x510E527F;    h[5]=0x9B05688C;    h[6]=0x1F83D9AB;    h[7]=0x5BE0CD19;

	__m128i row1, row2, row3, row4;
	__m128i buf1, buf2;

	union {
		uint32_t u32[16];
		__m128i u128[4];
	} m;
	static const int sig[][16] = {
		{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } , { 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 } ,
		{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 } , {  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 } ,
		{  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 } , {  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 } ,
		{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 } , { 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 } ,
		{  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 } , { 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 } ,
		{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } , { 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 } ,
		{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 } , {  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 }
	};
	static const uint32_t z[16] = {
		0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344, 0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C, 0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
	};
	/* get message */
	m.u128[0] = LOADU(endiandata + 0);
	m.u128[1] = LOADU(endiandata + 4);
	m.u128[2] = LOADU(endiandata + 8);
	m.u128[3] = LOADU(endiandata + 12);
	BSWAP32(m.u128[0]); BSWAP32(m.u128[1]); BSWAP32(m.u128[2]); BSWAP32(m.u128[3]);

	row1 = _mm_set_epi32(h[ 3], h[ 2], h[ 1], h[ 0]);
	row2 = _mm_set_epi32(h[ 7], h[ 6], h[ 5], h[ 4]);
	row3 = _mm_set_epi32(0x03707344, 0x13198A2E, 0x85A308D3, 0x243F6A88);
	row4 = _mm_set_epi32(0xEC4E6C89, 0x082EFA98, 0x299F31D0^512, 0xA4093822^512);

	round( 0);  round( 1);  round( 2);
	round( 3);  round( 4);  round( 5);
	round( 6);  round( 7);

	_mm_store_si128( (__m128i *)m.u32, _mm_xor_si128(row1,row3));
	h[0] ^= m.u32[ 0];  h[1] ^= m.u32[ 1];
	h[2] ^= m.u32[ 2];  h[3] ^= m.u32[ 3];
	_mm_store_si128( (__m128i *)m.u32, _mm_xor_si128(row2,row4));
	h[4] ^= m.u32[ 0];  h[5] ^= m.u32[ 1];
	h[6] ^= m.u32[ 2];  h[7] ^= m.u32[ 3];

	uint32_t tmp = h[5];
	h[ 5] = h[6];
	h[ 6] = h[7];
	h[ 7] = penddata[0];
	h[ 8] = penddata[1];
	h[ 9] = penddata[2];
	h[10] = SPH_C32(0xA4093822) ^ 640;
	h[11] = SPH_C32(0x243F6A88);

	h[ 0] += (h[7] ^ SPH_C32(0x85A308D3)) + h[4];
	h[10]  = SPH_ROTR32(h[10] ^ h[0],16);
	h[11] += h[10];
	h[ 4]  = SPH_ROTR32(h[4] ^ h[11], 12);
	h[ 0] += (h[8] ^ SPH_C32(0x243F6A88)) + h[4];
	h[10]  = SPH_ROTR32(h[10] ^ h[0],8);
	h[11] += h[10];
	h[ 4]  = SPH_ROTR32(h[4] ^ h[11], 7);

	h[1] += (h[ 9] ^ SPH_C32(0x03707344)) + tmp;

	h[12] = SPH_ROTR32(SPH_C32(0x299F31D0) ^ 640 ^ h[1],16);
	h[13] = ROTR32(tmp ^ (SPH_C32(0x85A308D3) + h[12]), 12);

	h[ 1] += h[13];
	h[ 2] += (0x80000000UL ^ SPH_C32(0x299F31D0)) + h[5];

	h[14]  = SPH_ROTR32(SPH_C32(0x082EFA98) ^ h[2], 16);
	h[15]  = SPH_C32(0x13198A2E) + h[14];
	h[15]  = SPH_ROTR32(h[5] ^ h[15], 12);

	h[ 3] += SPH_C32(0xEC4E6C89) + h[6];
	h[ 0] += SPH_C32(0x38D01377);

	h[ 2] += SPH_C32(0xA4093822) + h[15];

	hipMemcpyToSymbol(HIP_SYMBOL(d_data), h, 16*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_blake256(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, const int8_t blakerounds)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce  = pdata[19];
	const uint32_t targetHigh   = ptarget[6];
	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 30 : 24;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;

	if (!init[thr_id]) {
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		CUDA_CALL_OR_RET_X(hipHostAlloc((void**)&h_resNonce[thr_id], NBN*sizeof(uint32_t), hipHostMallocMapped),0);
		CUDA_CALL_OR_RET_X(hipHostGetDevicePointer((void**)&d_resNonce[thr_id],(void*)h_resNonce[thr_id], 0),0);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];

	for (int k = 0; k < 16; k++)
		be32enc(&endiandata[k], pdata[k]);

	vanilla_cpu_setBlock_16(endiandata,&pdata[16]);

	hipMemset(d_resNonce[thr_id], 0xff, sizeof(uint32_t));
	const dim3 grid((throughput + (NPT*TPB)-1)/(NPT*TPB));
	const dim3 block(TPB);
	do {
		vanilla_gpu_hash_16_8<<<grid,block>>>(throughput, pdata[19], d_resNonce[thr_id], targetHigh);
		hipDeviceSynchronize(); //Dont forget me. If you do, i'll probably read garbage on next instruction
		
		if (h_resNonce[thr_id][0] != UINT32_MAX){
			uint32_t vhashcpu[8];
			uint32_t Htarg = (uint32_t)targetHigh;

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], h_resNonce[thr_id][0]);
			blake256hash(vhashcpu, endiandata, blakerounds);

			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)){
				rc = 1;
				work_set_target_ratio(work, vhashcpu);
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = h_resNonce[thr_id][0];
#if NBN > 1
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					pdata[21] = h_resNonce[thr_id][1];
					applog(LOG_BLUE, "1:%x 2:%x", h_resNonce[thr_id][0], h_resNonce[thr_id][1]);
					rc = 2;
				}
#endif
				return rc;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[thr_id][0]);
			}
		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	MyStreamSynchronize(NULL, 0, dev_id);
	return rc;
}

// cleanup
extern "C" void free_blake256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
