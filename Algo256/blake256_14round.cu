#include "hip/hip_runtime.h"
/**
 * 14-round Blake-256 Cuda Kernel (Tested on SM 5.2) for SaffronCoin
 * Provos Alexis - April 2016
 *
 * Based on blake256 ccminer implementation of
 * Tanguy Pruvot / SP - Jan 2016
 *
 *
 * April 2016: +9% speed increase: 1396Mh/s -> 1522Mh/s on GTX970 at 1252MHz
 */

#include <stdint.h>
#include <memory.h>

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
}

/* threads per block */
#define TPB 640
/* max count of found nonces in one call */
#define NBN 1

/* hash by cpu with blake 256 */
extern "C" void blake256_14roundHash(void *output, const void *input)
{
	uchar hash[64];
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);

	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

__constant__ uint32_t _ALIGN(16) c_v[16];
__constant__ uint32_t c_h[ 2];
__constant__ uint32_t c_x[39];

/* 8 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

#define GS(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ z[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c],12); \
	v[a] += (m[y] ^ z[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
	}

#define hostX(x,y){\
	xors[i++] = x ^ y; \
}
#define hostGS(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ z[y]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ z[x]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
}
	
/* ############################################################################################################################### */
/* Precalculated 1st 64-bytes block (midstate) method */

__global__
void blake256_14round_gpu_hash_16(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t highTarget)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		const uint32_t m4 = 0x80000000U;
		const uint32_t z[16] = {
			0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,	0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,	0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
			0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
		};

		uint32_t v[16];
		#pragma unroll
		for(size_t i = 0; i < 16; i++)
			v[i] = c_v[i];


		v[ 1]+= (nonce ^ z[ 2]) + v[ 5];
		v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);
		v[ 9]+= v[13];
		v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		
		int i=0;
		
		v[ 0]+=          z[ 9]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 8]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[11]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[10]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
							v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= (1U ^ z[12])	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[15]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (640U ^ z[14])	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[10]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[14]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+= (m4    ^ z[ 8]) + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+=          z[ 4]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[15]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= (640U ^ z[ 9])	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= (1U ^ z[ 6])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[13]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 1]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+= c_x[i++]	+ v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+= c_x[i++] 	+ v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 7]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+=          z[11]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[ 3]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (nonce ^ z[ 5]) + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 8]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[11]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[ 0]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= c_x[i++] 	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[ 2]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= c_x[i++] 	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= (640U ^ z[13])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+= (1U ^ z[15])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+=          z[14]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[10]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+= (nonce ^ z[ 6]) + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[ 3]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 1]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= c_x[i++]	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[ 4]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (m4    ^ z[ 9]) + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 9]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[ 7]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+= (nonce ^ z[ 1]) + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= c_x[i++]	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+= (1U ^ z[12])	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+=          z[13]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+=          z[14]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[11]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 2]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[10]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[ 5]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+= (m4    ^ z[ 0]) + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= c_x[i++]	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+= (640U ^ z[ 8])	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+=          z[15]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 0]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+= c_x[i++]	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[ 7]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+=          z[ 5]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+= c_x[i++]	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= (m4    ^ z[ 2]) + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+=          z[15]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+= (640U ^ z[10])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+=          z[ 1]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[12]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[11]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 8]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+=          z[ 6]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+= (nonce ^ z[13]) + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (1U ^ z[ 3])	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+= c_x[i++]	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[ 2]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[10]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+=          z[ 6]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+= c_x[i++]	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+=          z[ 0]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+=          z[ 3]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+= (nonce ^ z[ 8]) + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= (m4    ^ z[13]) + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+= (1U ^ z[ 4])	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[ 5]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[ 7]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+= (640U ^ z[14])	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+=          z[15]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+= c_x[i++]	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+=          z[ 1]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 5]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[12]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+= c_x[i++]	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= (640U ^ z[ 1])	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[13]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= (1U ^ z[14])	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= (m4    ^ z[10]) + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[ 4]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 0]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[ 3]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+= (nonce ^ z[ 6]) + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 2]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= c_x[i++]	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[11]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+=          z[ 8]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+= (1U ^ z[11])	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[13]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[14]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+=          z[ 7]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[ 1]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= c_x[i++]	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= (nonce ^ z[ 9]) + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[ 3]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+=          z[ 0]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+= (640U ^ z[ 4])	+ v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+= (m4    ^ z[15]) + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 6]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+=          z[ 8]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+= c_x[i++]	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+=          z[ 2]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[15]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+= (640U ^ z[ 6])	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[ 9]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+=          z[14]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[ 3]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= (nonce ^ z[11]) + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= c_x[i++]	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[ 0]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+=          z[ 2]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+= (1U ^ z[ 7])	+ v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[13]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+= c_x[i++]	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= (m4    ^ z[ 1]) + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[ 5]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+=          z[10]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 2]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+= c_x[i++]	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[ 4]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= (m4    ^ z[ 8]) + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[ 6]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+=          z[ 7]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= c_x[i++]	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[ 1]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= (640U ^ z[11])	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[15]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[14]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[ 9]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+= (nonce ^ z[12]) + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+=          z[ 3]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+= (1U ^ z[ 0])	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= c_x[i++] 	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+= c_x[i++]	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+= c_x[i++]	+ v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+= c_x[i++]	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= (nonce ^ z[ 2]) + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+= (m4    ^ z[ 5]) + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+=          z[ 4]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+=          z[ 7]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[ 6]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+=          z[ 9]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 8]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[11]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[10]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[13]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= (1U ^ z[12])	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[15]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (640U ^ z[14])	+ v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[10]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[14]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+= (m4    ^ z[ 8]) + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+=          z[ 4]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[15]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= (640U ^ z[ 9])	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= (1U ^ z[ 6])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[13]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= c_x[i++]	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 1]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+= c_x[i++]	+ v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+= c_x[i++]	+ v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 7]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+=          z[11]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[ 3]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (nonce ^ z[ 5]) + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 8]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[11]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+=          z[ 0]  + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= c_x[i++]	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+=          z[ 2]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+= c_x[i++]	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+= (640U ^ z[13])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+= (1U ^ z[15])	+ v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+=          z[14]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[10]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+= (nonce ^ z[ 6]) + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[ 3]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+=          z[ 1]  + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= c_x[i++] 	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);
		v[ 3]+=          z[ 4]  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
		v[ 3]+= (m4    ^ z[ 9]) + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
		v[ 0]+=          z[ 9]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x1032);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8],12);
		v[ 0]+=          z[ 7]  + v[ 4];	v[12] = __byte_perm(v[12] ^ v[ 0],0, 0x0321);v[ 8]+= v[12];v[ 4] = ROTR32(v[ 4] ^ v[ 8], 7);
		v[ 1]+= (nonce ^ z[ 1]) + v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x1032);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
		v[ 1]+= c_x[i++] 	+ v[ 5];	v[13] = __byte_perm(v[13] ^ v[ 1],0, 0x0321);v[ 9]+= v[13];v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);
		v[ 2]+= (1U ^ z[12]) 	+ v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x1032);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10],12);
		v[ 2]+=          z[13]  + v[ 6];	v[14] = __byte_perm(v[14] ^ v[ 2],0, 0x0321);v[10]+= v[14];v[ 6] = ROTR32(v[ 6] ^ v[10], 7);
		v[ 3]+=          z[14]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x1032);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11],12);
		v[ 3]+=          z[11]  + v[ 7];	v[15] = __byte_perm(v[15] ^ v[ 3],0, 0x0321);v[11]+= v[15];v[ 7] = ROTR32(v[ 7] ^ v[11], 7);
		v[ 0]+= c_x[i++] 	+ v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x1032);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10],12);
		v[ 0]+=          z[ 2]  + v[ 5];	v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);v[10]+= v[15];v[ 5] = ROTR32(v[ 5] ^ v[10], 7);
		v[ 1]+=          z[10]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x1032);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11],12);
		v[ 1]+=          z[ 5]  + v[ 6];	v[12] = __byte_perm(v[12] ^ v[ 1],0, 0x0321);v[11]+= v[12];v[ 6] = ROTR32(v[ 6] ^ v[11], 7);
		v[ 2]+= (m4    ^ z[ 0]) + v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x1032);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8],12);
		v[ 2]+= c_x[i++] 	+ v[ 7];	v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);v[ 8]+= v[13];v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);

		// only compute h6 & 7
		if ((v[15]^c_h[ 1]) == v[ 7]){
			v[ 3]+=  (640U^z[ 8])  + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x1032);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9],12);
			v[ 3]+=      z[15] + v[ 4];	v[14] = __byte_perm(v[14] ^ v[ 3],0, 0x0321);v[ 9]+= v[14];v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);
			if(cuda_swab32(xor3x(v[ 6],c_h[ 0],v[14])) <= highTarget) {
#if NBN == 2
				if (resNonce[0] != UINT32_MAX)
					resNonce[1] = nonce;
				else
					resNonce[0] = nonce;
#else
				resNonce[0] = nonce;
#endif
			}
		}
	}
}

__host__
void blake256_14round_cpu_setBlock_16(const uint32_t *pendd, const uint32_t *input){

	const uint32_t z[16] = {
		0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
	};
		
	uint32_t _ALIGN(64) v[16];
	sph_blake256_context ctx;
	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 64);
	//memcpy(data, (void*)ctx.H, 32);
	v[ 0] = ctx.H[ 0];
	v[ 1] = ctx.H[ 1];
	v[ 2] = ctx.H[ 2];
	v[ 3] = ctx.H[ 3];
	v[ 4] = ctx.H[ 4];
	v[ 5] = ctx.H[ 5];
	v[ 6] = ctx.H[ 6];
	v[ 7] = ctx.H[ 7];
	v[ 8] = z[ 0];
	v[ 9] = z[ 1];
	v[10] = z[ 2];
	v[11] = z[ 3];
	v[12] = z[ 4] ^ 640;
	v[13] = z[ 5] ^ 640;
	v[14] = z[ 6];
	v[15] = z[ 7];	

	const uint32_t h[2]	= { 	v[ 6],		v[ 7]};
	
	const uint32_t m[16] 	= { 	pendd[ 0],	pendd[ 1], pendd[ 2],	0,
					0x80000000,	0,		0,		0,
					0,		0,		0,		0,
					0,		1,		0,		640
				};

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_h), h, 2*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	
	hostGS(	0, 4, 8,12, 0, 1);	hostGS(	2, 6,10,14, 4, 5);	hostGS(	3, 7,11,15, 6, 7);
	
	v[ 1]+= (m[ 2] ^ z[ 3]) + v[ 5];
	v[13] = ROTR32(v[13] ^ v[ 1],16);
	v[ 9] += v[13];
	v[ 5] = ROTR32(v[ 5] ^ v[ 9],12);
	v[ 2]+= z[13]  + v[ 7];
	
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_v), v, 16*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	
	int i=0;
	uint32_t xors[39];
	xors[i++] = (m[ 1] ^ z[12]);	xors[i++] = (m[ 0] ^ z[ 2]);	xors[i++] = (m[ 2] ^ z[ 0]);	xors[i++] = (m[ 0] ^ z[12]);
	xors[i++] = (m[ 2] ^ z[ 5]);	xors[i++] = (m[ 1] ^ z[ 7]);	xors[i++] = (m[ 1] ^ z[ 3]);	xors[i++] = (m[ 2] ^ z[ 6]);
	xors[i++] = (m[ 0] ^ z[ 4]);	xors[i++] = (m[ 0] ^ z[ 9]);	xors[i++] = (m[ 2] ^ z[ 4]);	xors[i++] = (m[ 1] ^ z[14]);
	xors[i++] = (m[ 2] ^ z[12]);	xors[i++] = (m[ 0] ^ z[11]);	xors[i++] = (m[ 1] ^ z[ 9]);	xors[i++] = (m[ 1] ^ z[15]);
	xors[i++] = (m[ 0] ^ z[ 7]);	xors[i++] = (m[ 2] ^ z[ 9]);	xors[i++] = (m[ 1] ^ z[12]);	xors[i++] = (m[ 0] ^ z[ 5]);
	xors[i++] = (m[ 2] ^ z[10]);	xors[i++] = (m[ 0] ^ z[ 8]);	xors[i++] = (m[ 2] ^ z[12]);	xors[i++] = (m[ 1] ^ z[ 4]);
	xors[i++] = (m[ 2] ^ z[10]);	xors[i++] = (m[ 1] ^ z[ 5]);	xors[i++] = (m[ 0] ^ z[13]);	xors[i++] = (m[ 0] ^ z[ 1]);
	xors[i++] = (m[ 1] ^ z[ 0]);	xors[i++] = (m[ 2] ^ z[ 3]);	xors[i++] = (m[ 1] ^ z[12]);	xors[i++] = (m[ 0] ^ z[ 2]);
	xors[i++] = (m[ 2] ^ z[ 0]);	xors[i++] = (m[ 0] ^ z[12]);	xors[i++] = (m[ 2] ^ z[ 5]);	xors[i++] = (m[ 1] ^ z[ 7]);
	xors[i++] = (m[ 1] ^ z[ 3]);	xors[i++] = (m[ 2] ^ z[ 6]);	xors[i++] = (m[ 0] ^ z[ 4]);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_x), xors, 39*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_blake256_14round(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];
	uint64_t targetHigh = ((uint64_t*)ptarget)[3];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 30 : 26;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	dim3 grid((throughput + TPB-1)/TPB);
	dim3 block(TPB);
	
	int rc = 0;

	if (opt_benchmark) {
		targetHigh = 0x1ULL << 32;
		ptarget[6] = swab32(0xff);
	}

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}

	for (int k = 0; k < 16; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_14round_cpu_setBlock_16(&pdata[16], endiandata);
	hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
	do {
		// GPU HASH (second block only, first is midstate)
		blake256_14round_gpu_hash_16  <<<grid, block>>> (throughput, pdata[19], d_resNonce[thr_id], targetHigh);
		hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);
		if (h_resNonce[thr_id][0] != UINT32_MAX){
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[6];

			for (int k=16; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], h_resNonce[thr_id][0]);
			blake256_14roundHash(vhashcpu, endiandata);

			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)){
				rc = 1;
				work_set_target_ratio(work, vhashcpu);
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = h_resNonce[thr_id][0];
#if NBN > 1
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					be32enc(&endiandata[19], h_resNonce[thr_id][1]);
					blake256_14roundHash(vhashcpu, endiandata);
					if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)) {
						pdata[21] = h_resNonce[thr_id][1];
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio) {
							work_set_target_ratio(work, vhashcpu);
							xchg(pdata[21], pdata[19]);
						}
						rc = 2;
					}
				}
#endif
				return rc;
			}
			else{
				applog_hash((uchar*)ptarget);
				applog_compare_hash((uchar*)vhashcpu, (uchar*)ptarget);
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[thr_id][0]);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + pdata[19]);

	*hashes_done = pdata[19] - first_nonce;

	MyStreamSynchronize(NULL, 0, device_map[thr_id]);
	return rc;
}

// cleanup
extern "C" void free_blake256_14round(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

